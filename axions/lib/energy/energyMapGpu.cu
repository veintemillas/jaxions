#include "hip/hip_runtime.h"
#include "complexGpu.cuh"
#include "index.cuh"

#include "enum-field.h"
#include "hipcub/hipcub.hpp"

#define	BLSIZE 512

using namespace gpuCu;
using namespace indexHelper;

__device__ uint bCount = 0;

template <int bSize, typename Float>
__device__ inline void reduction(Float * __restrict__ eRes, const Float * __restrict__ tmp, Float *partial)
{
	typedef hipcub::BlockReduce<Float, bSize, hipcub::BLOCK_REDUCE_WARP_REDUCTIONS> BlockReduce;
	const int blockSurf = gridDim.x*gridDim.y;

	__shared__ bool isLastBlockDone;
	__shared__ typename BlockReduce::TempStorage cub_tmp[10];
//	__shared__ typename BlockReduce::TempStorage cub_tmp;


	Float tmpGrhx = BlockReduce(cub_tmp[0]).Sum(tmp[0]);
	Float tmpGthx = BlockReduce(cub_tmp[1]).Sum(tmp[1]);
	Float tmpGrhy = BlockReduce(cub_tmp[2]).Sum(tmp[2]);
	Float tmpGthy = BlockReduce(cub_tmp[3]).Sum(tmp[3]);
	Float tmpGrhz = BlockReduce(cub_tmp[4]).Sum(tmp[4]);
	Float tmpGthz = BlockReduce(cub_tmp[5]).Sum(tmp[5]);
	Float tmpVrho = BlockReduce(cub_tmp[6]).Sum(tmp[6]);
	Float tmpVth  = BlockReduce(cub_tmp[7]).Sum(tmp[7]);
	Float tmpKrho = BlockReduce(cub_tmp[8]).Sum(tmp[8]);
	Float tmpKth  = BlockReduce(cub_tmp[9]).Sum(tmp[9]);

//	Float tmp = BlockReduce(cub_tmp).Sum(*tmpC);

	if (threadIdx.x == 0)
	{
		const int bIdx = blockIdx.x + gridDim.x*blockIdx.y;

		partial[bIdx + 0*blockSurf] = tmpGrhx;
		partial[bIdx + 1*blockSurf] = tmpGthx;
		partial[bIdx + 2*blockSurf] = tmpGrhy;
		partial[bIdx + 3*blockSurf] = tmpGthy;
		partial[bIdx + 4*blockSurf] = tmpGrhz;
		partial[bIdx + 5*blockSurf] = tmpGthz;
		partial[bIdx + 6*blockSurf] = tmpVrho;
		partial[bIdx + 7*blockSurf] = tmpVth;
		partial[bIdx + 8*blockSurf] = tmpKrho;
		partial[bIdx + 9*blockSurf] = tmpKth;

//		partial[bIdx] = tmp;

		__threadfence();

		unsigned int cBlock = atomicInc(&bCount, blockSurf);
		isLastBlockDone = (cBlock == (blockSurf-1));
	}

	__syncthreads();

	// finish the reduction if last block
	if (isLastBlockDone)
	{
		uint i = threadIdx.x;

		tmpGrhx = 0., tmpGthx = 0.;
		tmpGrhy = 0., tmpGthy = 0.;
		tmpGrhz = 0., tmpGthz = 0.;
		tmpVrho = 0., tmpVth  = 0.;
		tmpKrho = 0., tmpKth  = 0.;

//		tmp = 0.;

		while (i < blockSurf)
		{

			tmpGrhx += partial[i + 0*blockSurf];
			tmpGthx += partial[i + 1*blockSurf];
			tmpGrhy += partial[i + 2*blockSurf];
			tmpGthy += partial[i + 3*blockSurf];
			tmpGrhz += partial[i + 4*blockSurf];
			tmpGthz += partial[i + 5*blockSurf];
			tmpVrho += partial[i + 6*blockSurf];
			tmpVth  += partial[i + 7*blockSurf];
			tmpKrho += partial[i + 8*blockSurf];
			tmpKth  += partial[i + 9*blockSurf];

//			tmp  += partial[i];

			i += bSize;
		}

		tmpGrhx = BlockReduce(cub_tmp[0]).Sum(tmpGrhx);
		tmpGthx = BlockReduce(cub_tmp[1]).Sum(tmpGthx);
		tmpGrhy = BlockReduce(cub_tmp[2]).Sum(tmpGrhy);
		tmpGthy = BlockReduce(cub_tmp[3]).Sum(tmpGthy);
		tmpGrhz = BlockReduce(cub_tmp[4]).Sum(tmpGrhz);
		tmpGthz = BlockReduce(cub_tmp[5]).Sum(tmpGthz);
		tmpVrho = BlockReduce(cub_tmp[6]).Sum(tmpVrho);
		tmpVth  = BlockReduce(cub_tmp[7]).Sum(tmpVth);
		tmpKrho = BlockReduce(cub_tmp[8]).Sum(tmpKrho);
		tmpKth  = BlockReduce(cub_tmp[9]).Sum(tmpKth);

//		tmp = BlockReduce(cub_tmp).Sum(tmp);

		if (threadIdx.x == 0)
		{

			eRes[0] = tmpGrhx;
			eRes[1] = tmpGthx;
			eRes[2] = tmpGrhy;
			eRes[3] = tmpGthy;
			eRes[4] = tmpGrhz;
			eRes[5] = tmpGthz;
			eRes[6] = tmpVrho;
			eRes[7] = tmpVth;
			eRes[8] = tmpKrho;
			eRes[9] = tmpKth;

//			eRes[0] = tmp;

			bCount = 0;
		}
	}
}


template<typename Float>
static __device__ __forceinline__ void	energyCoreGpu(const uint idx, const complex<Float> * __restrict__ m, const complex<Float> * __restrict__ v, const uint Lx, const uint Sf, const double iZ, const double iZ2, double *tR)
{
	uint X[3], idxPx, idxPy, idxMx, idxMy;

	complex<Float> mDX, mDY, mDZ, tmp, vOm;

	idx2Vec(idx, X, Lx);

	if (X[0] == Lx-1)
		idxPx = idx - Lx+1;
	else
		idxPx = idx+1;

	if (X[0] == 0)
		idxMx = idx + Lx-1;
	else
		idxMx = idx-1;

	if (X[1] == Lx-1)
		idxPy = idx - Sf + Lx;
	else
		idxPy = idx + Lx;

	if (X[1] == 0)
		idxMy = idx + Sf - Lx;
	else
		idxMy = idx - Lx;

	tmp = m[idx];

	Float mod = tmp.real()*tmp.real() + tmp.imag()*tmp.imag();
	Float mFac = iZ2*mod;
	Float iMod = 1./mod;


	vOm = v[idx-Sf]*conj(m[idx])*iMod - gpuCu::complex<Float>(iZ, 0.);
	mDX = (m[idxPx]  - m[idxMx]) *conj(m[idx])*iMod;
	mDY = (m[idxPy]  - m[idxMy]) *conj(m[idx])*iMod;
	mDZ = (m[idx+Sf] - m[idx-Sf])*conj(m[idx])*iMod;


	tR[0] = (double) ((Float) (mFac*mDX.real()*mDX.real()));
	tR[1] = (double) ((Float) (mFac*mDX.imag()*mDX.imag()));
	tR[2] = (double) ((Float) (mFac*mDY.real()*mDY.real()));
	tR[3] = (double) ((Float) (mFac*mDY.imag()*mDY.imag()));
	tR[4] = (double) ((Float) (mFac*mDZ.real()*mDZ.real()));
	tR[5] = (double) ((Float) (mFac*mDZ.imag()*mDZ.imag()));
	tR[6] = (double) ((Float) (mFac - 1.)*(mFac - 1.));
	tR[7] = (double) (((Float) 1.) - tmp.real()*iZ);
	tR[8] = (double) ((Float) (mFac*vOm.real()*vOm.real()));
	tR[9] = (double) ((Float) (mFac*vOm.imag()*vOm.imag()));
}

template<typename Float>
__global__ void	energyKernel(const complex<Float> * __restrict__ m, const complex<Float> * __restrict__ v, const uint Lx, const uint Sf, const uint V, const double iZ, const double iZ2, double *eR, double *partial)
{
	uint idx = Sf + (threadIdx.x + blockDim.x*(blockIdx.x + gridDim.x*blockIdx.y));

	double tmp[10] = { 0., 0., 0., 0., 0., 0., 0., 0., 0., 0. };

	if	(idx < V)
		energyCoreGpu<Float>(idx, m, v, Lx, Sf, iZ, iZ2, tmp);

	reduction<BLSIZE,double>   (eR, tmp, partial);
}

int	energyGpu	(const void * __restrict__ m, const void * __restrict__ v, double *z, const double delta2, const double LL, const double nQcd,
			 const uint Lx, const uint Lz, const uint V, const uint Vt, const uint S, FieldPrecision precision, double *eR, hipStream_t &stream)
{
	const uint Vm = V+S;
	const uint Lz2 = V/(Lx*Lx);
	dim3  gridSize((Lx*Lx+BLSIZE-1)/BLSIZE,Lz2,1);
	dim3  blockSize(BLSIZE,1,1);
	const int nBlocks = gridSize.x*gridSize.y;

	const double zR   = *z;
	double *tR, *partial;

	if ((hipMalloc(&tR, sizeof(double)*10) != hipSuccess) || (hipMalloc(&partial, sizeof(double)*10*nBlocks*4) != hipSuccess))
	{
		return -1;
	}

	if (precision == FIELD_DOUBLE)
	{
		const double iZ  = 1./zR;
		const double iZ2 = iZ*iZ;
		energyKernel<<<gridSize,blockSize,0,stream>>> (static_cast<const complex<double>*>(m), static_cast<const complex<double>*>(v), Lx, S, Vm, iZ, iZ2, tR, partial);
	}
	else if (precision == FIELD_SINGLE)
	{
		const float iZ = 1./zR;
		const float iZ2 = iZ*iZ;
		energyKernel<<<gridSize,blockSize,0,stream>>> (static_cast<const complex<float>*>(m), static_cast<const complex<float>*>(v), Lx, S, Vm, iZ, iZ2, tR, partial);
	}

	hipDeviceSynchronize();

	hipMemcpy(eR, tR, sizeof(double)*10, hipMemcpyDeviceToHost);
	hipFree(tR); hipFree(partial);

	const double iV = 1./((double) Vt);
	const double o2 = 0.375/delta2;
	const double zQ = 9.*pow(zR, nQcd+2.);
	const double lZ = 0.25*LL*zR*zR;

	eR[0] *= o2*iV;
	eR[1] *= o2*iV;
	eR[2] *= o2*iV;
	eR[3] *= o2*iV;
	eR[4] *= o2*iV;
	eR[5] *= o2*iV;
	eR[6] *= lZ*iV;
	eR[7] *= zQ*iV;
	eR[8] *= .5*iV;
	eR[9] *= .5*iV;

	return	0;
}
