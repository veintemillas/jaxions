#include "hip/hip_runtime.h"
#include "complexGpu.cuh"
#include "utils/index.cuh"

#include "enum-field.h"
#include "hipcub/hipcub.hpp"

#define	BLSIZE 512

using namespace gpuCu;
using namespace indexHelper;


template<typename Float>
static __device__ __forceinline__ int	stringHand(const complex<Float> s1, const complex<Float> s2)
{
	int hand = 0;

	if (s1.imag()*s2.imag() < 0)
		hand = (((s1*conj(s2)).imag() > 0)<<1) - 1;

	return hand;
}


template<typename Float>
static __device__ __forceinline__ void	stringCoreGpu(const uint idx, const complex<Float> * __restrict__ m, const uint Lx, const uint Sf, void * __restrict__ str)
{
	uint X[3], idxPx, idxPy, idxXY, idxYZ, idxZX;

	complex<Float> mel, mPx, mXY, mPy, mYZ, mPz, mZX;
	uint sIdx = idx-Sf;
	int hand = 0;
	char strDf = 0;

	idx2Vec(idx, X, Lx);

	if (X[0] == Lx-1)
	{
		idxPx = idx - Lx + 1;
		idxZX = idxPx + Sf;

		if (X[1] == Lx-1)
		{
			idxPy = sIdx + Lx;
			idxXY = sIdx + 1;
			idxYZ = idx + Lx;
		} else {
			idxPy = idx + Lx;
			idxXY = idx + 1;
			idxYZ = idx + Sf + Lx;
		}
	} else {
		idxPx = idx + 1;
		idxZX = idxPx + Sf;

		if (X[1] == Lx-1)
		{
			idxPy = sIdx + Lx;
			idxYZ = idx + Lx;
		} else {
			idxPy = idx + Lx;
			idxYZ = idx + Sf + Lx;
		}

		idxXY = idxPy + 1;
	}

	mel = m[idx];
	mPx = m[idxPx];
	mPy = m[idxPy];
	mXY = m[idxXY];
	mPz = m[idx+Sf];
	mZX = m[idxZX];
	mYZ = m[idxYZ];

	// Primera plaqueta XY

	hand += stringHand (mel, mPx);
	hand += stringHand (mPx, mXY);
	hand += stringHand (mXY, mPy);
	hand += stringHand (mPy, mel);
/*	ARREGLAR PARA QUE SOLO HAYA UN STORE	*/
/*	LA QUIRALIDAD SE GUARDA MAL	*/
	if (hand == 2)
		strDf |= STRING_XY_POSITIVE;
	else if (hand == -2)
		strDf |= STRING_XY_NEGATIVE;

	hand = 0;

	// Segunda plaqueta YZ

	hand += stringHand (mel, mPy);
	hand += stringHand (mPy, mYZ);
	hand += stringHand (mYZ, mPz);
	hand += stringHand (mPz, mel);

	if (hand == 2)
		strDf |= STRING_YZ_POSITIVE;
	else if (hand == -2)
		strDf |= STRING_YZ_NEGATIVE;

	hand = 0;

	// Tercera plaqueta ZX

	hand += stringHand (mel, mPz);
	hand += stringHand (mPz, mZX);
	hand += stringHand (mZX, mPx);
	hand += stringHand (mPx, mel);

	if (hand == 2)
		strDf |= STRING_ZX_POSITIVE;
	else if (hand == -2)
		strDf |= STRING_ZX_NEGATIVE;

	static_cast<char *>(str)[sIdx] = strDf;
}

template<typename Float>
__global__ void	stringKernel(void * __restrict__ strg, const complex<Float> * __restrict__ m, const uint Lx, const uint Sf, const uint V)
{
	uint idx = Sf + (threadIdx.x + blockDim.x*(blockIdx.x + gridDim.x*blockIdx.y));

	if	(idx < V)
		stringCoreGpu<Float>(idx, m, Lx, Sf, strg);
}

double	stringGpu	(const void * __restrict__ m, const uint Lx, const uint V, const uint S, FieldPrecision precision, void * __restrict__ str, hipStream_t &stream)
{
	const uint Vm = V+S;
	const uint Lz2 = V/(Lx*Lx);
	dim3  gridSize((Lx*Lx+BLSIZE-1)/BLSIZE,Lz2,1);
	dim3  blockSize(BLSIZE,1,1);

	void   *strg;

	if (hipMalloc(&strg, sizeof(char)*V) != hipSuccess)
		return -1;

	if (precision == FIELD_DOUBLE)
		stringKernel<<<gridSize,blockSize,0,stream>>> (strg, static_cast<const complex<double>*>(m), Lx, S, Vm);
	else if (precision == FIELD_SINGLE)
		stringKernel<<<gridSize,blockSize,0,stream>>> (strg, static_cast<const complex<float>*>(m), Lx, S, Vm);

	hipDeviceSynchronize();

	hipMemcpy(str, strg, sizeof(char)*V, hipMemcpyDeviceToHost);
	hipFree(strg);

	return	0;
}
