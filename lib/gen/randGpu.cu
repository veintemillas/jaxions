#include "hip/hip_runtime.h"
#include <random>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "complexGpu.cuh"
#include "scalar/scalarField.h"
#include "enum-field.h"

#include "comms/comms.h"

#define	BLSIZE	512
#define	CTSIZE	262144

using namespace gpuCu;

__global__ void	randInitGpu (hiprandState_t * state, const uint seed, const uint rank, const uint size)
{
	uint bIdx = blockIdx.x + gridDim.x*blockIdx.y;
	uint idx  = threadIdx.x + blockDim.x*bIdx;

	hiprand_init (seed*gridDim.x*gridDim.y + rank*size*gridDim.x*gridDim.y + bIdx, threadIdx.x, 0, &state[idx]);
}

template<typename Float>
__global__ void	randKernelGpu (hiprandState_t * __restrict__ state, complex<Float> * __restrict__ m, const uint Vo, const uint Vf, const uint workPerThread)
{
	uint size = Vf - Vo;
	uint idx  = threadIdx.x + blockDim.x*(blockIdx.x + gridDim.x*blockIdx.y);
	uint wIdx = idx*workPerThread;

	if	(wIdx < size) {

		hiprandState_t localState = state[idx];

		#pragma unroll
		for	(int k=0; k<workPerThread; k++) {

			Float	rP = (Float) (2.*hiprand_uniform (&localState) - 1.);
			Float	iP = (Float) (2.*hiprand_uniform (&localState) - 1.);

			m[Vo + wIdx + k]	= complex<Float>(rP, iP);
		}

		state[idx]	= localState;
	}
}

void	randGpu (Scalar *field)
{
	const uint	S  = field->Surf();
	const uint	V  = field->Size();
	const uint	Lz = field->Depth();

	size_t		memGpu = gpuMemAvail();

	hipStream_t	&stream = static_cast<hipStream_t *>(field->Streams())[0];

	if	(field->LowMem())
		memGpu -= (2*V+field->nGhost())*field->DataSize();
	else
		memGpu -= (3*V+2*field->nGhost())*field->DataSize();

	memGpu	*= 3;		// El factor 0.75 es para dejar hueco por si las moscas
	memGpu	/= 4*sizeof(hiprandState_t);

	if	(memGpu > V)
		memGpu = V;

	printf	("Allocating %lu bytes for the pRNG\n", memGpu*sizeof(hiprandState_t));
	fflush	(stdout);

	hiprandState_t	*state;
	if ((hipMalloc(&state, memGpu*sizeof(hiprandState_t))) != hipSuccess) {
		printf("Error: Couldn't allocate %zu bytes in device for random number generator\n", memGpu*sizeof(hiprandState_t));
		exit (1);
	}

	printf("\nParallel RNG using %zu bytes in device\n", memGpu*sizeof(hiprandState_t));
	fflush(stdout);

	std::random_device seed;

	dim3		gridSize((memGpu/Lz+BLSIZE-1)/BLSIZE,Lz,1);
	dim3		blockSize(BLSIZE,1,1);

	randInitGpu  <<<gridSize,blockSize,0,stream>>>(state, seed(), commRank(), memGpu);

	uint	workPerThread = S/(memGpu/Lz);

	printf("GridSize\t(%d %d %d)\nBlockSize\t(%d %d %d)\nWork per Thread %d\n", gridSize.x, gridSize.y, gridSize.z, blockSize.x, blockSize.y, blockSize.z, workPerThread);
	fflush(stdout);

	switch (field->Precision())
	{
		case FIELD_DOUBLE:
		//randKernelGpu<<<gridSize,blockSize,0,stream>>>(state, static_cast<complex<double>*> (field->mGpu()), S, V+S, workPerThread);
		break;

		case FIELD_SINGLE:
		//randKernelGpu<<<gridSize,blockSize,0,stream>>>(state, static_cast<complex<float> *> (field->mGpu()), S, V+S, workPerThread);
		break;

		default:
		break;
	}

	hipFree (state);
}

#undef	BLSIZE
